#include "hip/hip_runtime.h"
#pragma once
#include "Header.h"
#define DEFULT_RADIUS 1000000
int getNumberOfBlocks(int size);
void error(Point * p);
void error(Point * p, float *r);

/*the global function insert to radius array the min value she found */
__global__ void internalRadius(int size, Point *points, float * radius)
{
	int i;
	int b = blockIdx.x;
	int t = threadIdx.x;
	int index = b*BLOCK_SIZE + t;
	for (i = index + 1; i < size; i++)
	{
		if (points[index].clusterId == points[i].clusterId)
		{
			float temp = sqrt(pow(points[index].x - points[i].x, 2) + pow(points[index].y - points[i].y, 2));
			if (temp > radius[index])
				radius[index] = temp;
		}
	}
}

/*the global function insert to radius array  0 */
__global__ void initRadius(int size, float * radius)
{
	int index = blockIdx.x*BLOCK_SIZE + threadIdx.x;
	if (index < size)
		radius[index] = 0;
}
hipError_t internalRadiusWithCuda(int size, Point *points, float *radius)
{
	int numberOfBlocks = 0;
	Point *dev_points = 0;
	float *dev_radius = 0;
	hipError_t cudaStatus;
	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "internalRadiusWithCuda - hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		fflush(stdout);
		error(points, radius);
	}

	//  cuda malloc   .
	cudaStatus = hipMalloc((void**)&dev_points, size * sizeof(Point));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "internalRadiusWithCuda -  dev_points hipMalloc failed!");
		fflush(stdout);
		error(points, radius);
	}
	cudaStatus = hipMalloc((void**)&dev_radius, size * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "internalRadiusWithCuda -  dev_radius hipMalloc failed!");
		fflush(stdout);
		error(points, radius);
	}

	
	numberOfBlocks = getNumberOfBlocks(size);
	
	// hipMemcpyHostToDevice.
	cudaStatus = hipMemcpy(dev_points, points, size * sizeof(Point), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "internalRadiusWithCuda - dev_points hipMemcpy failed!(hipMemcpyHostToDevice)");
		error(points, radius);
	}

	// insert values to dev_radius.
	internalRadius <<<numberOfBlocks, BLOCK_SIZE >>> (size, dev_points, dev_radius);

	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		printf( "internalRadiusWithCuda launch failed (internalRadius): %s\n");
		error(points, radius);
	}
	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching movePoints!\n", cudaStatus);
		error(points);

	}
	cudaStatus = hipMemcpy(radius, dev_radius, size * sizeof(float),
		hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "internalRadiusWithCuda - hipMemcpy failed!(hipMemcpyDeviceToHost)");
		error(points, radius);
	}
	error(points, radius);

	return cudaStatus;
}

/*the global function move the points  */
__global__ void movePoints(int size, Point *points, double Dtime)
{
	int b = blockIdx.x;
	int t = threadIdx.x;
	int index = b*BLOCK_SIZE + t;
	if (index < size)
	{
		points[index].x += Dtime*points[index].speedX;
		points[index].y += Dtime*points[index].speedY;
	}
}
hipError_t movePointsWithCuda(int numberOfPoints, Point *points, double Dtime)
{
	int numberOfBlocks = 0;
	Point *dev_points = 0;
	hipError_t cudaStatus;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "movePointsWithCuda - hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		error(points);
	}
	// Allocate GPU buffers for three vectors (two input, one output)    .
	cudaStatus = hipMalloc((void**)&dev_points, numberOfPoints * sizeof(Point));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "movePointsWithCuda - hipMalloc failed!\n");
		error(points);

	}
	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_points, points, numberOfPoints * sizeof(Point), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "movePointsWithCuda - dev_points hipMemcpy failed! (hipMemcpyHostToDevice)\n");
		error(points);

	}

	numberOfBlocks = getNumberOfBlocks(numberOfPoints);
	movePoints  <<<numberOfBlocks, BLOCK_SIZE >>> (numberOfPoints, dev_points, Dtime);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "movePoints launch failed (movePoints): %s\n", hipGetErrorString(cudaStatus));
		error(points);

	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching movePoints!\n", cudaStatus);
		error(points);

	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(points, dev_points, numberOfPoints * sizeof(Point),
		hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "movePointsWithCuda - hipMemcpy failed!(hipMemcpyDeviceToHost)");
		error(points);

	}
	//cudaStatus = hipDeviceReset();
	//if (cudaStatus != hipSuccess)
	//{
	//	fprintf(stderr, "hipDeviceReset failed!");
	//	error(points);
	//}
	error(points);
	return cudaStatus;
}
int getNumberOfBlocks(int size)
{
	int num = size / BLOCK_SIZE;
	if (size % BLOCK_SIZE > 0 || num == 0) num++;
	return num;
}
void error(Point * p)
{
	hipFree(p);
}
void error(Point * p, float *r)
{
	hipFree(p);
	hipFree(r);
}
