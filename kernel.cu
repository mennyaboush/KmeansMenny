#include "hip/hip_runtime.h"

#include "Header.h"
int getNumberOfBlocks(int size);
void error(Point * p);

//hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);



__global__ void movePoints(int size, Point *points , int Dtime)
{
	int b = blockIdx.x;
	int t = threadIdx.x;
	int index = b*BLOCK_SIZE + t;
	if (index < size)
	{
		points[index].x += Dtime*points[index].speedX;
		points[index].y += Dtime*points[index].speedY;
	}
}

hipError_t movePointsWithCuda(int size, Point *points , int Dtime)
{
	int numberOfBlocks = 0;
	Point *dev_points = 0;
	hipError_t cudaStatus;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "movePointsWithCuda - hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		error(points);
	}
	// Allocate GPU buffers for three vectors (two input, one output)    .
	cudaStatus = hipMalloc((void**)&dev_points, size * sizeof(Point));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "movePointsWithCuda - hipMalloc failed!");
		error(points);

	}
	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_points, points, size * sizeof(Point), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "movePointsWithCuda - dev_points hipMemcpy failed!");
		error(points);

	}

	numberOfBlocks = getNumberOfBlocks(size); 
	movePoints <<<numberOfBlocks, BLOCK_SIZE>>>(size, points, Dtime);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "movePoints launch failed: %s\n", hipGetErrorString(cudaStatus));
		error(points);

	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching movePoints!\n", cudaStatus);
		error(points);

	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(points, dev_points, size * sizeof(Point), 
		hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "movePointsWithCuda - hipMemcpy failed!");
		error(points);

	}
	return cudaStatus;
}

int getNumberOfBlocks(int size)
{
	int num = size / BLOCK_SIZE;
	if (size % BLOCK_SIZE > 0 || num == 0) num++;
	return num;
}

void error(Point * p)
{
	hipFree(p);
}
